
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void hello_from_gpu() {
    printf("Hello from the GPU!\\n");
}

int main() {
    std::cout << "Hello from the CPU!" << std::endl;
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
